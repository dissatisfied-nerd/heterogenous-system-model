#include "hip/hip_runtime.h"
#include "kernel/GPU.cuh"
#include <chrono>

__global__ void matrixMulKernel(const double* A, const double* B, double* C,
    int aRows, int aCols, int bCols) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < aRows && col < bCols) 
    {
        double sum = 0.0;
        for (int k = 0; k < aCols; ++k) {
            sum += A[row * aCols + k] * B[k * bCols + col];
        }
        C[row * bCols + col] = sum;
    }
}

std::pair<double, double> multiplyGPU(const std::vector<std::vector<double>>& a,
    const std::vector<std::vector<double>>& b,
    BufferManager& bufferManager) 
{
    int aRows = a.size();
    int aCols = a[0].size();
    int bCols = b[0].size();

    std::vector<double> A(aRows * aCols);
    std::vector<double> B(b.size() * bCols);
    std::vector<double> C(aRows * bCols);

    for (int i = 0; i < aRows; ++i)
        for (int j = 0; j < aCols; ++j)
            A[i * aCols + j] = a[i][j];

    for (int i = 0; i < (int)b.size(); ++i)
        for (int j = 0; j < bCols; ++j)
            B[i * bCols + j] = b[i][j];

    // Получаем буферы из менеджера
    auto& buf = bufferManager.getBuffers(aRows, aCols, bCols);

    // Events
    hipEvent_t startTransfer, endTransfer, startCompute, endCompute;
    hipEventCreate(&startTransfer);
    hipEventCreate(&endTransfer);
    hipEventCreate(&startCompute);
    hipEventCreate(&endCompute);

    hipEventRecord(startTransfer);
    hipMemcpy(buf.d_A, A.data(), buf.aSize * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(buf.d_B, B.data(), buf.bSize * sizeof(double), hipMemcpyHostToDevice);
    hipEventRecord(endTransfer);

    dim3 blockDim(16, 16);
    dim3 gridDim((bCols + 15) / 16, (aRows + 15) / 16);
    hipEventRecord(startCompute);
    matrixMulKernel<<<gridDim, blockDim>>>(buf.d_A, buf.d_B, buf.d_C, aRows, aCols, bCols);
    hipEventRecord(endCompute);

    hipMemcpy(C.data(), buf.d_C, buf.cSize * sizeof(double), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();

    float transferTime = 0.0f, computeTime = 0.0f;
    hipEventElapsedTime(&transferTime, startTransfer, endTransfer);
    hipEventElapsedTime(&computeTime, startCompute, endCompute);

    hipEventDestroy(startTransfer);
    hipEventDestroy(endTransfer);
    hipEventDestroy(startCompute);
    hipEventDestroy(endCompute);

    return {computeTime, transferTime};
}

std::pair<double, double> multiplyGPU_async(const std::vector<std::vector<double>>& a,
    const std::vector<std::vector<double>>& b,
    BufferManager& bufferManager) 
{
    int aRows = a.size();
    int aCols = a[0].size();
    int bCols = b[0].size();

    std::vector<double> A(aRows * aCols);
    std::vector<double> B(b.size() * bCols);
    std::vector<double> C(aRows * bCols);

    for (int i = 0; i < aRows; ++i)
    for (int j = 0; j < aCols; ++j)
    A[i * aCols + j] = a[i][j];

    for (int i = 0; i < (int)b.size(); ++i)
    for (int j = 0; j < bCols; ++j)
    B[i * bCols + j] = b[i][j];

    auto& buf = bufferManager.getBuffers(aRows, aCols, bCols);

    hipStream_t stream;
    hipStreamCreate(&stream);

    hipEvent_t startTransfer, endTransfer, startCompute, endCompute;
    hipEventCreate(&startTransfer);
    hipEventCreate(&endTransfer);
    hipEventCreate(&startCompute);
    hipEventCreate(&endCompute);

    hipEventRecord(startTransfer, stream);
    hipMemcpyAsync(buf.d_A, A.data(), buf.aSize * sizeof(double), hipMemcpyHostToDevice, stream);
    hipMemcpyAsync(buf.d_B, B.data(), buf.bSize * sizeof(double), hipMemcpyHostToDevice, stream);
    hipEventRecord(endTransfer, stream);

    dim3 blockDim(16, 16);
    dim3 gridDim((bCols + 15) / 16, (aRows + 15) / 16);
    hipEventRecord(startCompute, stream);
    matrixMulKernel<<<gridDim, blockDim, 0, stream>>>(buf.d_A, buf.d_B, buf.d_C, aRows, aCols, bCols);
    hipEventRecord(endCompute, stream);

    hipMemcpyAsync(C.data(), buf.d_C, buf.cSize * sizeof(double), hipMemcpyDeviceToHost, stream);

    hipStreamSynchronize(stream);

    float transferTime = 0.0f, computeTime = 0.0f;
    hipEventElapsedTime(&transferTime, startTransfer, endTransfer);
    hipEventElapsedTime(&computeTime, startCompute, endCompute);

    hipEventDestroy(startTransfer);
    hipEventDestroy(endTransfer);
    hipEventDestroy(startCompute);
    hipEventDestroy(endCompute);
    hipStreamDestroy(stream);

    return {computeTime, transferTime};
}

