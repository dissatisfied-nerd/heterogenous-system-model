#include "memory/GPUPool.cuh"
#include <stdexcept>

GPUMemoryPool::GPUMemoryPool() {
    hipDeviceGetDefaultMemPool(&memPool, 0);
}

GPUMemoryPool::~GPUMemoryPool() {}

void* GPUMemoryPool::mallocAsync(size_t size, hipStream_t stream) 
{
    void* ptr = nullptr;
    hipMallocAsync(&ptr, size, stream);

    if (!ptr) {
        throw std::runtime_error("Failed to hipMallocAsync");
    }

    return ptr;
}

void GPUMemoryPool::freeAsync(void* ptr, hipStream_t stream) {
    hipFreeAsync(ptr, stream);
}
